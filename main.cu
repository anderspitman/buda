#include "hip/hip_runtime.h"
#include <iostream>
#include <quickbam/slicer.h>
#include <quickbam/mbgzf.h>

__global__ void hithere();

int main(int argc, char** argv) {
    auto bam_path = argv[1];
    auto bai_path = std::string(bam_path) + ".bai";

    file_slicer_t bam_slicer(bam_path);

    auto bgzf_it = bgzf_slicer_iterator_t<file_slicer_t>(bam_slicer);
    auto bgzf_end = bgzf_it.end();

    while (bgzf_it != bgzf_end) {
        //std::cout << "Hi there" << std::endl;
        bgzf_it++;
    }

    hithere<<<1,1>>>();
    hipDeviceSynchronize();
}
